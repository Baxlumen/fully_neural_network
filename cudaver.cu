#include "hip/hip_runtime.h"
#include "head.h"

__global__ void Vector2_Multiply_By_Elements (const double* a, const double* b, int n, double* out){
	int tid = blockIdx.x*blockDim.x + threadIdx.x;
	const long temp = blockDim.x*gridDim.x;
	while(tid<n)
	{
		out[tid]=a[tid]*(1.0-a[tid])*(b[tid]-a[tid]);
		tid+=temp;
	}
	__syncthreads();
}
__global__ void Vector1_Multiply_By_Elements (const double* a, double* b, int n){
	int tid = blockIdx.x*blockDim.x + threadIdx.x;
	const long temp = blockDim.x*gridDim.x;
	while(tid<n)
	{
		b[tid]=a[tid]*(1.0-a[tid])*b[tid];
		tid+=temp;
	}
	__syncthreads();
}

__global__ void activate(double *d_A,int b,int a)
{
	int tid = blockIdx.x*blockDim.x + threadIdx.x;
	const long temp = blockDim.x*gridDim.x;
	while(tid<a)
	{
		d_A[tid] = 1/(1+exp(-d_A[tid]+b));
		tid+=temp;
	}
	__syncthreads();
}
void printTensor(double *d_des,long m,long n,long l){
	double *des = new double[m*n*l]();
	hipMemcpy(des,d_des,sizeof(double)*m*n*l,hipMemcpyDeviceToHost);
	hipDeviceSynchronize();
	for(long k = 0;k<l;k++){
		for(long i = 0;i<n;i++){
			for(long j = 0;j<m;j++){
				cout<<des[k*m*n+i*m+j]<<" ";
			}
			cout<<endl;
		}
		cout<<"~~~~~~~~~~~~~~~~"<<endl;
	}
	delete[] des;des=nullptr;
}

void for_cuda(double *input,double *W1,double *outh,double *W2,double *outo,int in,int hid,int out,hipblasHandle_t handle)
{
	//hid 行 in列 W1 ,out行，hid列 W2

	//printTensor(input,5,1,1);
	//cout<<"weigh matrix is :"<<endl;printTensor(W1,4,4,1);

	//double *outo;
	//hipMalloc((void**)&outo,sizeof(double)*out);
	int b= (rand() % 100) / (double)100; //偏置值
	double alpha=1.0, beta=0.0;


	hipblasDgemm(handle,HIPBLAS_OP_N,HIPBLAS_OP_N,hid,1,in,&alpha,W1,hid,input,in,&beta,outh,hid);

	//激活函数
	activate<<<1,1024>>>(outh,b,hid);
	//printTensor(outh,3,3,1);

	hipDeviceSynchronize();

	hipblasDgemm(handle,HIPBLAS_OP_N,HIPBLAS_OP_N,out,1,hid,&alpha,W2,out,outh,hid,&beta,outo,out);	
	activate<<<1,1024>>>(outo,b,out);
	hipDeviceSynchronize();
	//cout<<"model output:"<<endl;printTensor(outo,2,2,1);
}

void back_cuda(double *Y,double *Y_hat,double *outh,double *W2,double *input,double *W1,int in,int hid,int out,double rate,hipblasHandle_t handle)
{
	//cout<<"yu ce value:"<<endl;printTensor(Y_hat,10,1,1);
	//cout<<"bp zhong de W1"<<endl;printTensor(W1,4,4,1);
	double *d_thta3,*d_thta2;
	hipMalloc((void**)&d_thta3,sizeof(double)*out);
	hipMalloc((void**)&d_thta2,sizeof(double)*hid);
	Vector2_Multiply_By_Elements<<<1,512>>>(Y_hat, Y, out, d_thta3);
	double alpha=1.0, beta=0.0;
	hipblasDgemm(handle,HIPBLAS_OP_T,HIPBLAS_OP_N,hid,1,out,&alpha,W2,out,d_thta3,out,&beta,d_thta2,hid);
	Vector1_Multiply_By_Elements<<<1,512>>>(outh, d_thta2, hid);

	alpha=rate; beta=1.0;
	hipblasDgemm(handle,HIPBLAS_OP_N,HIPBLAS_OP_N,hid,in,1,&alpha,d_thta2,hid,input,1,&beta,W1,hid);
	hipblasDgemm(handle,HIPBLAS_OP_N,HIPBLAS_OP_N,out,hid,1,&alpha,d_thta3,out,outh,1,&beta,W2,out);

	//printTensor(W1,4,4,1);

	hipFree(d_thta2);
	hipFree(d_thta3);
}	
double loss_gpu(double *A,double *B,int n,hipblasHandle_t handle)
{	
	//A 实际值  B预测值
	//printTensor(A,2,2,1);
	//printTensor(B,2,2,1);
	double alpha1 = -1.0,loss;
	double *tmp;
	hipMalloc((void**)&tmp,sizeof(double)*n);

	hipblasDcopy(handle,n,B,1,tmp,1);	
	hipblasDaxpy(handle,n,&alpha1,A,1,tmp,1);
	hipblasDnrm2(handle,n,tmp,1,&loss); 
	hipDeviceSynchronize();
	hipFree(tmp);
	return loss;
}

